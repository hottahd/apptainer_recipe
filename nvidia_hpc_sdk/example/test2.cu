#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
    } else {
        printf("CUDA-capable devices found: %d\n", deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);

            printf("Device %d: %s\n", i, prop.name);
            printf("  Total Global Memory: %lu bytes\n", prop.totalGlobalMem);
            printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
            printf("  CUDA Cores (estimated): %d\n", prop.multiProcessorCount * 128);
        }
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Warp size: %d\n", prop.warpSize);
    printf("Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);

    return 0;
}