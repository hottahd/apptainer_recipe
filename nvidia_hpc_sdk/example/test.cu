#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDAカーネル: 配列aとbを足し合わせて結果を配列cに格納
__global__ void add_arrays(const float *a, const float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // グローバルインデックス
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 1024; // 配列のサイズ
    size_t size = n * sizeof(float);

    // ホストメモリの確保
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // デバイスメモリのポインタ
    float *d_a, *d_b, *d_c;

    // デバイスメモリの確保
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // ホスト配列を初期化
    for (int i = 0; i < n; i++) {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
    }

    // ホストメモリからデバイスメモリへコピー
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // ブロックとグリッドのサイズを設定
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // nをブロックサイズで割り切る

    // カーネルの呼び出し
    add_arrays<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // デバイスメモリからホストメモリへ結果をコピー
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // 結果の表示
    printf("Result:\n");
    for (int i = 0; i < 10; i++) { // 最初の10個を表示
        printf("h_c[%d] = %f\n", i, h_c[i]);
    }

    // メモリの解放
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}