
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_kernel() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    hello_kernel<<<1, 2048>>>();
    hipDeviceSynchronize();
    return 0;
}