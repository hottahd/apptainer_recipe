// device_query.cu

#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << static_cast<int>(error_id)
                  << " -> " << hipGetErrorString(error_id) << std::endl;
        return 1;
    }

    std::cout << "Detected " << device_count << " CUDA capable device(s)." << std::endl;

    for (int dev = 0; dev < device_count; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << ": " << deviceProp.name << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Multi-processor count: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Warp size: " << deviceProp.warpSize << std::endl;
    }

    return 0;
}