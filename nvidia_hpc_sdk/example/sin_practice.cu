
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

__global__ void fill_sin_kernel(double *data, double *x, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = (sin(x[idx + 1]) - sin(x[idx - 1]))/(x[idx + 1] - x[idx - 1]);
    }
}

int main() {
    const int N = 1024;
    const double xmax = 2.0 * M_PI;
    const double dx = xmax / N;

    // geometry
    double *d_x = nullptr;
    double *h_x = new double[N];

    for (int i = 0; i < N; ++i) {
        h_x[i] = i * dx;
    };
    
    // data
    double *d_data = nullptr;
    double *h_data = new double[N];

    hipMalloc(&d_x   , N * sizeof(double));
    hipMalloc(&d_data, N * sizeof(double));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipMemcpy(d_x, h_x   , N * sizeof(double), hipMemcpyHostToDevice);
    fill_sin_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_x, N);
    hipDeviceSynchronize();
    hipMemcpy(h_data, d_data, N * sizeof(double), hipMemcpyDeviceToHost);

    printf("First 10 values of sin(x):\n");
    for (int i = 0; i < 1024; i++ ) {
        printf("%d, cos: %.6f, cos:%.6f\n", i, cos(h_x[i]), h_data[i]);
    }

    hipFree(d_data);
    delete[] h_data;

    return 0;
}   